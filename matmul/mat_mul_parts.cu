#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/errors.h"

#define N 10000
#define P 1000

__global__ void add(int *a, int *b, int *c) {
    int bid = blockIdx.x;
	int tid = bid * 256 + threadIdx.x;
	if (tid < N) {
        for(int i=0; i<100; i++){
            c[tid] = a[tid] + b[tid];
        }
    }
}

int main(void) {
	int a[N], b[N], c[N];
	int *devA, *devB, *devC;

	HANDLE_ERROR(hipMalloc((void**)&devA, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devB, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devC, N * sizeof(int)));

	//fill a and b arrays
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

    for(int i=0; i<N; i+=P){
        HANDLE_ERROR(hipMemcpy(devA + i, a + i, P * sizeof(int), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(devB + i, b + i, P * sizeof(int), hipMemcpyHostToDevice));
        add<<<(P+255)/256,256>>>(devA, devB, devC);
        HANDLE_ERROR(hipMemcpy(c, devC + i, P * sizeof(int), hipMemcpyDeviceToHost));
    }

	
	//check if ok
    for(int i=0; i<N; i++){
        if(c[i] != a[i] + b[i]){
            printf("error at cell %d, %d != %d\n", i, c[i], a[i]+b[i]);
            return 1;
        }
    }

    printf("success\n");

	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));
	return 0;
}