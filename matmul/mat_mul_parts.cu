#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/errors.h"

#define N 10000
#define P 1000

__global__ void add(int *a, int *b, int *c) {
    int bid = blockIdx.x;
	int tid = bid * 256 + threadIdx.x;
	if (tid < N) {
        for(int i=0; i<1000; i++){
            c[tid] = a[tid] + b[tid];
        }
    }
}

int main(void) {

    hipStream_t stream[10];
    for(int i=0; i<10; i++){
        hipStreamCreate(stream + i);
    }
	int a[N], b[N], c[N];
	int *devA, *devB, *devC;

	HANDLE_ERROR(hipMalloc((void**)&devA, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devB, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devC, N * sizeof(int)));

	//fill a and b arrays
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

    for(int j=0; j<10; j++){
        int i = j * P
        printf("copying indexes %d to %d", i, i+P);
        HANDLE_ERROR(hipMemcpyAsync(devA + i, a + i, P * sizeof(int), hipMemcpyHostToDevice, stream[j]));
        HANDLE_ERROR(hipMemcpyAsync(devB + i, b + i, P * sizeof(int), hipMemcpyHostToDevice, stream[j]));
        add<<<(P+255)/256,256, 0, stream[j]>>>(devA+i, devB +i, devC + i);
        HANDLE_ERROR(hipMemcpyAsync(c+i, devC + i, P * sizeof(int), hipMemcpyDeviceToHost,stream[j]));
    }

    for(int i=0; i<10; i++){
        hipStreamSynchronize(stream[j]);
    }
	
	//check if ok
    for(int i=0; i<N; i++){
        if(c[i] != a[i] + b[i]){
            printf("error at cell %d, %d != %d\n", i, c[i], a[i]+b[i]);
            return 1;
        }
    }

    printf("success\n");

	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));


    for(int i=0; i<10; i++){
        hipStreamDestroy(stream + i);
    }
	return 0;
}