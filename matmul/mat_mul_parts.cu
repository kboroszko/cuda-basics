#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/errors.h"

#define N 10000
#define P 1000

__global__ void add(int *a, int *b, int *c) {
    int bid = blockIdx.x;
	int tid = bid * 256 + threadIdx.x;
	if (tid < N) {
        for(int i=0; i<1000; i++){
            c[tid] = a[tid] + b[tid];
        }
    }
}

int main(void) {

    hipStream_t stream;
    hipStreamCreate(&stream);

	int a[N], b[N], c[N];
	int *devA, *devB, *devC;

	HANDLE_ERROR(hipMalloc((void**)&devA, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devB, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devC, N * sizeof(int)));

	//fill a and b arrays
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

    for(int i=0; i<N; i+=P){
        printf("copying indexes %d to %d", i, i+P);
        HANDLE_ERROR(hipMemcpyAsync(devA + i, a + i, P * sizeof(int), hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(devB + i, b + i, P * sizeof(int), hipMemcpyHostToDevice, stream));
        add<<<(P+255)/256,256, 0, stream>>>(devA+i, devB +i, devC + i);
        HANDLE_ERROR(hipMemcpyAsync(c+i, devC + i, P * sizeof(int), hipMemcpyDeviceToHost,stream));
    }

    hipStreamSynchronize(stream);
	
	//check if ok
    for(int i=0; i<N; i++){
        if(c[i] != a[i] + b[i]){
            printf("error at cell %d, %d != %d\n", i, c[i], a[i]+b[i]);
            return 1;
        }
    }

    printf("success\n");

	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));


    hipStreamDestroy(stream);
	return 0;
}