#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/errors.h"

#define N 1000

__global__ void add(int *a, int *b, int *c) {
    int bid = blockIdx.x;
	int tid = bid * 256 + threadIdx.x;
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
    }
}

int main(void) {
	int a[N], b[N], c[N];
	int *devA, *devB, *devC;

	HANDLE_ERROR(hipMalloc((void**)&devA, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devB, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devC, N * sizeof(int)));

	//fill a and b arrays
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

	HANDLE_ERROR(hipMemcpy(devA, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERRORcudaMemcpy(devB, b, N * sizeof(int), hipMemcpyHostToDevice));
	add<<<(N+255)/256,256>>>(devA, devB, devC);
	HANDLE_ERRORcudaMemcpy(c, devC, N * sizeof(int), hipMemcpyDeviceToHost));


	//check if ok
    for(int i=0; i<N; i++){
        if(c[i] != a[i] + b[i]){
            printf("error at cell %d, %d != %d", i, c[i], a[i]+b[i]);
            return 1;
        }
    }

	HANDLE_ERRORcudaFree(devA));
	HANDLE_ERRORcudaFree(devB));
	HANDLE_ERRORcudaFree(devC));
	return 0;
}