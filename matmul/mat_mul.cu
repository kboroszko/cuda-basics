#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common/errors.h"

#define N 10000

__global__ void add(int *a, int *b, int *c) {
    int bid = blockIdx.x;
	int tid = bid * 256 + threadIdx.x;
	if (tid < N) {
        for(int i=0; i<100; i++){
            c[tid] = a[tid] + b[tid];
        }
    }
}

int main(void) {
	int a[N], b[N], c[N];
	int *devA, *devB, *devC;

	HANDLE_ERROR(hipMalloc((void**)&devA, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devB, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&devC, N * sizeof(int)));

	//fill a and b arrays
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

	HANDLE_ERROR(hipMemcpy(devA, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devB, b, N * sizeof(int), hipMemcpyHostToDevice));
	add<<<(N+255)/256,256>>>(devA, devB, devC);
	HANDLE_ERROR(hipMemcpy(c, devC, N * sizeof(int), hipMemcpyDeviceToHost));


	//check if ok
    for(int i=0; i<N; i++){
        if(c[i] != a[i] + b[i]){
            printf("error at cell %d, %d != %d\n", i, c[i], a[i]+b[i]);
            return 1;
        }
    }

    printf("success\n");

	HANDLE_ERROR(hipFree(devA));
	HANDLE_ERROR(hipFree(devB));
	HANDLE_ERROR(hipFree(devC));
	return 0;
}