
#include <hip/hip_runtime.h>
__constant__ float constants[3];

__global__ void someKernel() {
    //code that reads from constants
}

int main(void) {
    float *cpuConstants = (float*)malloc(sizeof(float) * 3);
    cpuConstants[0] = 9.81f;
    cpuConstants[1] = 3.14f;
    cpuConstants[2] = 1.66f;

    hipMemcpyToSymbol(HIP_SYMBOL(constants), cpuConstants, sizeof(float) * 3);

    free(cpuConstants);

    someKernel<<<1,1>>>();
        
    return 0;
}