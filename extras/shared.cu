
#include <hip/hip_runtime.h>
__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (tid < size) {
            atomicAdd(&temp[buffer[tid]], 1);
            tid += stride;
    }
    __syncthreads();

    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}