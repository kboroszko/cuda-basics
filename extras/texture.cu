#include "hip/hip_runtime.h"
#define DIM 1024

texture<float, 2> simpleTexture;

__global__ void textureReadKernel() {

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;  
    
    float top, left, center, right, bottom;
    top = tex2D(simpleTexture, x, y - 1);    
    left = tex2D(simpleTexture, x - 1, y);
    center = tex2D(simpleTexture, x, y);
    right = tex2D(simpleTexture, x + 1, y);
    bottom = tex2D(simpleTexture, x, y + 1);

    //do something with top, left, center, right, bottom
}

int main(void) {
    dim3 blocks(DIM / 16, DIM / 16);
    dim3 threads(16, 16);
    float *devicePointer;
    float *hostPointer = (float*)malloc(SIZE * sizeof(float));
    hipMalloc((void**)&devicePointer, SIZE * sizeof(float));

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(NULL, simpleTexture, devicePointer, desc, SIZE, SIZE, SIZE * sizeof(float));

    //fill hostPointer with data

    hipMemcpy(devicePointer, hostPointer, SIZE * sizeof(float), hipMemcpyHostToDevice);

    textureReadKernel<<<blocks, threads>>>();
    
    hipUnbindTexture(simpleTexture);
    hipFree(devicePointer);
    free(hostPointer);
}