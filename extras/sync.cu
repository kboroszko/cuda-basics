#include "hip/hip_runtime.h"
const int N = 30 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = min(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {

    __shared__ float cache[threadsPerBlock];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;
    
    __syncthreads();

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) { 
            cache[cacheIndex] += cache[cacheIndex + i]; 
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}

int main(void) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    //memory allocation and vector initialisation
    //host -> device data transfer

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    c = 0;
    for (int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    //free memory on device and on host

   return 0;
}       